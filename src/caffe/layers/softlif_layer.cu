#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include <vector>

#include "caffe/layers/softlif_layer.hpp"

namespace caffe {

__global__ const float tau_ref = 0.004;
__global__ const float tau_rc = 0.02;
__global__ const float v_th = 1.0; //voltage threshold
__global__ const float y = 0.5; //smoothing parameter

template <typename Dtype>
__global__ void SoftLIFForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = 1. / (tau_ref + tau_rc * log(1. + v_th/(y*log(1. + exp((in[index]-v_th)/y)))));
  }
}

template <typename Dtype>
void SoftLIFLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  SoftLIFForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

template <typename Dtype>
__global__ void SoftLIFBackward(const int n, const Dtype* in_diff,
    const Dtype* out_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    const Dtype x = out_data[index] - v_th;
    out_diff[index] = in_diff[index] * (tau_rc * v_th * exp(x/y) / (y*y*(exp(x/y)+1.) * log(exp(x/y)+1.) * (v_th/(y*log(exp(x/y)+1.))+1.) * pow((tau_rc*log(v_th/(y*log(exp(x/y)+1.))+1.) + tau_ref),2)));
  }
}

template <typename Dtype>
void SoftLIFLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    SoftLIFBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SoftLIFLayer);


}  // namespace caffe
